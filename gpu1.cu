#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include ""
#include "gpu1.cuh"
#include "HostConstants.h"

#include <stdio.h>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_vector.h>

#define DEBUG

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__global__ void CalculateBelongings(const float* clusters, const float* vectors, int* belonging, const int& N, const int& D, const int& K)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx >= N)
        return;

    int min_cluster = 0;
    float min_distance = FLT_MAX;

    for (int i = 0; i < K; i++) {
        float distance = 0.0f;
        for (int j = 0; j < D; j++) {
            float diff = vectors[idx + j * N] - clusters[i + j * K];
            distance += diff * diff;
        }
        if (distance < min_distance) {
            min_distance = distance;
            min_cluster = i;
        }
    }

    for (int i = 0; i < D; i++)
    {
        belonging[idx + i * N] = i * K + min_cluster;
    }
}

__global__ void CalculateClusters(float* clusters, const int* cluster_count, const int& D, const int& K)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = 0; i < D; i++)
    {
        clusters[idx + K * i] /= cluster_count[idx];
    }
}

void CalculateKmean(float* clusters, const float* vectors, int* belonging, int N, int K, int D)
{
    float* dev_clusters = 0;
    float* dev_vectors = 0;
    int* dev_belonging = 0;
    int* dev_n = 0;
    int* dev_k = 0;
    int* dev_d = 0;
    int* dev_cluster_count = 0;

    gpuErrchk(hipSetDevice(0));

    // Memory allocation on the side of the device
    gpuErrchk(hipMalloc((void**)&dev_clusters, K * D * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_vectors, N * D * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_belonging, N * D * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_n, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_k, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_d, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_cluster_count, K * sizeof(int)));

    // Copying memory from host to device
    gpuErrchk(hipMemcpy(dev_vectors, vectors, N * D * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_clusters, clusters, K * D * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_belonging, 0, N * D * sizeof(int)));
    gpuErrchk(hipMemcpy(dev_n, &N, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_k, &K, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_d, &D, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_cluster_count, 0, K * sizeof(int)));

    // pointers initialization
    thrust::device_ptr<int> keys(dev_belonging);
    thrust::device_ptr<float> vals(dev_vectors);
    thrust::device_ptr<float> clusters_ptr(dev_clusters);
    thrust::device_ptr<int> cluster_count_ptr(dev_cluster_count);
    thrust::constant_iterator<int> const_iter(1);

    //-------------------------------
    //            LOGIC
    //-------------------------------

    int block_count = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    CalculateBelongings << <block_count, THREADS_PER_BLOCK >> > (dev_clusters, dev_vectors, dev_belonging, *dev_n, *dev_d, *dev_k);

    // sorting 
    thrust::sort_by_key(keys, keys + N * D, vals);

    // reduction
    thrust::equal_to<int> binary_pred;
    thrust::reduce_by_key(keys, keys + N * D, vals, thrust::make_discard_iterator(), clusters_ptr, binary_pred);
    
    // updating clusters
    thrust::reduce_by_key(keys, keys + N, const_iter, thrust::make_discard_iterator(), cluster_count_ptr, binary_pred);
    CalculateClusters << <1, K>> > (dev_clusters, dev_cluster_count, *dev_d, *dev_k);

    //-------------------------------
    //         END OF LOGIC
    //-------------------------------

    // error checking and synchronization
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());


    // Copy memory back to the host
    gpuErrchk(hipMemcpy(clusters, dev_clusters, K * D * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(belonging, dev_belonging, N * D * sizeof(int), hipMemcpyDeviceToHost));
    
    // Cleaning
    gpuErrchk(hipFree(dev_clusters));
    gpuErrchk(hipFree(dev_vectors));
    gpuErrchk(hipFree(dev_belonging));
    gpuErrchk(hipFree(dev_n));
    gpuErrchk(hipFree(dev_k));
    gpuErrchk(hipFree(dev_d));
}