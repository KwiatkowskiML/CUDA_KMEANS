#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gpu1.cuh"

hipError_t CalculateKmean(float* clusters, const float* vectors, int* belonging, const int& N, const int& K, const int& D)
{
    float* dev_clusters = 0;
    float* dev_vectors = 0;
    float* dev_belonging = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_clusters, K * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_clusters!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_vectors, N * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_vectors!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_belonging, N * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_belonging!");
        goto Error;
    }



Error:
    hipFree(dev_clusters);
    hipFree(dev_vectors);
    hipFree(dev_belonging);

    return cudaStatus;
}