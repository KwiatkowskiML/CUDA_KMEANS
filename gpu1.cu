#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gpu1.cuh"

hipError_t CalculateKmean(float* clusters, const float* vectors, int* belonging, const int& N, const int& K, const int& D)
{
    float* dev_clusters = 0;
    float* dev_vectors = 0;
    int* dev_belonging = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_clusters, K * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_clusters!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_vectors, N * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_vectors!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_belonging, N * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_belonging!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_vectors, vectors, N * D * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for dev_vectors!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_clusters, dev_vectors, K * D * sizeof(float), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for copying dev_vectors to dev_clusters!");
        goto Error;
    } 

    cudaStatus = hipMemset(dev_belonging, 1, N * D * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed for dev_belonging!");
        goto Error;
    }

    cudaStatus = hipMemcpy(clusters, dev_clusters, K * D * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for clusters!");
        goto Error;
    }

    cudaStatus = hipMemcpy(belonging, dev_belonging, N * D * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for belonging!");
        goto Error;
    }

Error:
    hipFree(dev_clusters);
    hipFree(dev_vectors);
    hipFree(dev_belonging);

    return cudaStatus;
}