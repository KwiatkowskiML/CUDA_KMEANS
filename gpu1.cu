#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gpu1.cuh"

hipError_t CalculateKmean(float* clusters, const float* vectors, int* belonging, const int& N, const int& K, const int& D)
{
    float* dev_clusters = 0;
    float* dev_vectors = 0;
    int* dev_belonging = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    // Memory allocation on the side of the device
    cudaStatus = hipMalloc((void**)&dev_clusters, K * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_clusters!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_vectors, N * D * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_vectors!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_belonging, N * D * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for dev_belonging!");
        goto Error;
    }

    // Copying memory from host to device
    cudaStatus = hipMemcpy(dev_vectors, vectors, N * D * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for dev_vectors!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_clusters, clusters, K * D * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for to dev_clusters!");
        goto Error;
    } 

    cudaStatus = hipMemset(dev_belonging, 0, N * D * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed for dev_belonging!");
        goto Error;
    }

    //-------------------------------
    //            LOGIC
    //-------------------------------



    //-------------------------------
    //         END OF LOGIC
    //-------------------------------


    // Copy memory back to the host
    cudaStatus = hipMemcpy(clusters, dev_clusters, K * D * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for clusters!");
        goto Error;
    }

    cudaStatus = hipMemcpy(belonging, dev_belonging, N * D * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for belonging!");
        goto Error;
    }

Error:
    hipFree(dev_clusters);
    hipFree(dev_vectors);
    hipFree(dev_belonging);

    return cudaStatus;
}

__global__ void CalculateBelongings(const float* clusters, const float* vectors, int* belonging, const int& N, const int& D, const int& K)
{

}