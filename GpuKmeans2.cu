#include "hip/hip_runtime.h"
#include "GpuKmeans2.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sort.h>


__global__ void CalculateBelongings2(const float* clusters, const float* vectors, int* belonging, const int& N, const int& D, const int& K)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= N)
        return;

    int min_cluster = 0;
    float min_distance = FLT_MAX;

    for (int i = 0; i < K; i++) {
        float distance = 0.0f;
        for (int j = 0; j < D; j++) {
            float diff = vectors[idx + j * N] - clusters[i + j * K];
            distance += diff * diff;
        }
        if (distance < min_distance) {
            min_distance = distance;
            min_cluster = i;
        }
    }

    for (int i = 0; i < D; i++)
    {
        belonging[idx + i * N] = i * K + min_cluster;
    }
}

__global__ void CalculateClusters2(float* clusters, const int* cluster_count, const int& D, const int& K)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = 0; i < D; i++)
    {
        clusters[idx + K * i] /= cluster_count[idx];
    }
}

void GpuKmeans2::CalculateKmeans()
{
    int N = vectorsStorage->getNumPoints();
    int D = vectorsStorage->getNumDimensions();
    int K = vectorsStorage->getNumClusters();
    float* vectors = vectorsStorage->vectors;
    float* clusters = vectorsStorage->clusters;
    int* belonging = vectorsStorage->belonging;

    float* dev_clusters = 0;
    float* dev_vectors = 0;
    int* dev_belonging = 0;
    int* dev_n = 0;
    int* dev_k = 0;
    int* dev_d = 0;
    int* dev_cluster_count = 0;
    thrust::device_vector<int> vector_order(N);

    //-------------------------------
    //      TIME MEASUREMENT
    //-------------------------------

    float milliseconds = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //-------------------------------
    //      DATA PREPARATION
    //-------------------------------

    gpuErrchk(hipSetDevice(0));

    // Memory allocation on the side of the device
    gpuErrchk(hipMalloc((void**)&dev_clusters, K * D * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_vectors, N * D * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&dev_belonging, N * D * sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_n, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_k, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_d, sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_cluster_count, K * sizeof(int)));

    // Copying memory from host to device
    gpuErrchk(hipMemcpy(dev_vectors, vectors, N * D * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_clusters, clusters, K * D * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_belonging, 0, N * D * sizeof(int)));
    gpuErrchk(hipMemcpy(dev_n, &N, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_k, &K, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_d, &D, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(dev_cluster_count, 0, K * sizeof(int)));

    // pointers initialization
    thrust::device_ptr<int> keys(dev_belonging);
    thrust::device_ptr<float> vals(dev_vectors);
    thrust::device_ptr<float> clusters_ptr(dev_clusters);
    thrust::device_ptr<int> cluster_count_ptr(dev_cluster_count);
    thrust::constant_iterator<int> const_iter(1);
    thrust::counting_iterator<int> count_iter(0);
    thrust::equal_to<int> binary_pred;
    thrust::copy(count_iter, count_iter + N, vector_order.begin());

    //-------------------------------
    //            LOGIC
    //-------------------------------

    // calculating belonigng for each vector
    gpuErrchk(hipEventRecord(start, 0));
    int block_count = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    CalculateBelongings2 << <block_count, THREADS_PER_BLOCK >> > (dev_clusters, dev_vectors, dev_belonging, *dev_n, *dev_d, *dev_k);
    calculateElapsedTime(start, stop, &milliseconds, "Calculate belongings");

    // sorting the order of vectors, accordingly to the current order of belonging
    gpuErrchk(hipEventRecord(start, 0));
    thrust::sort_by_key(keys, keys + N, thrust::make_zip_iterator(thrust::make_tuple(vector_order.begin(), vals)));
    calculateElapsedTime(start, stop, &milliseconds, "Sorting vector order");

    // sorting the rest of the belongings
    gpuErrchk(hipEventRecord(start, 0));
    thrust::sort_by_key(keys + N, keys + N * D, vals + N);
    calculateElapsedTime(start, stop, &milliseconds, "Sorting belongings");

    // summing vectors in each cluster
    gpuErrchk(hipEventRecord(start, 0));
    thrust::reduce_by_key(keys, keys + N * D, vals, thrust::make_discard_iterator(), clusters_ptr, binary_pred);
    calculateElapsedTime(start, stop, &milliseconds, "Summing vectors in each cluster");

    // counting number of vectors in each cluster
    gpuErrchk(hipEventRecord(start, 0));
    thrust::reduce_by_key(keys, keys + N, const_iter, thrust::make_discard_iterator(), cluster_count_ptr, binary_pred);
    calculateElapsedTime(start, stop, &milliseconds, "Counting number of vectors in each cluster");

    // Updating cluster means
    gpuErrchk(hipEventRecord(start, 0));
    CalculateClusters2 << <1, K >> > (dev_clusters, dev_cluster_count, *dev_d, *dev_k);
    calculateElapsedTime(start, stop, &milliseconds, "Calculating means");

    //-------------------------------
    //         END OF LOGIC
    //-------------------------------

    // back to original vectors order
    gpuErrchk(hipEventRecord(start, 0));
    thrust::sort_by_key(vector_order.begin(), vector_order.end(), keys);
    calculateElapsedTime(start, stop, &milliseconds, "Resorting the belongings to original order");

    // error checking and synchronization
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    // Copy memory back to the host
    gpuErrchk(hipMemcpy(clusters, dev_clusters, K * D * sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(belonging, dev_belonging, N * sizeof(int), hipMemcpyDeviceToHost));

    // Cleaning
    gpuErrchk(hipFree(dev_clusters));
    gpuErrchk(hipFree(dev_vectors));
    gpuErrchk(hipFree(dev_belonging));
    gpuErrchk(hipFree(dev_n));
    gpuErrchk(hipFree(dev_k));
    gpuErrchk(hipFree(dev_d));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
}